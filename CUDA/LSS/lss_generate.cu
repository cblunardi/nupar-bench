#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include "image.h"

#define BUF_SIZE 256

using namespace std;

class errorPNM { };

struct Color
{
	unsigned char r;
	unsigned char g;
	unsigned char b;
};

void readPNM(ifstream &file, char* buf);
image<unsigned char>* loadPGM(const char* name);
void savePPM(image<Color>* im, const char* name);
Color randomColor();

__global__ void evolveContour(unsigned char* intensityDev, unsigned char* labelsDev, signed char* speedDev, signed char* phiDev, int HEIGHT, int WIDTH, int* targetLabels, int kernelID, int numLabels, int* lowerIntensityBounds, int* upperIntensityBounds);

__global__ void initSpeedPhi(unsigned char* intensity, unsigned char* labels, signed char* speed, signed char* phi, int HEIGHT, int WIDTH, int targetLabel, int lowerIntensityBound, int upperIntensityBound);

__global__ void switchIn(signed char* speed, signed char* phi, int HEIGHT, int WIDTH);
__global__ void switchOut(signed char* speed, signed char* phi, int HEIGHT, int WIDTH);

__global__ void checkStopCondition(signed char* speed, signed char* phi, int parentThreadID, int HEIGHT, int WIDTH);
__device__ volatile int stopCondition[1024];

void usage()
{
	cout<<"Usage: ./lss <Input intensities path> <Input labels path> <Input params path> <GOLD output path> <#repetitions (HyperQ)>" << endl;
}

int main(int argc, char* argv[])
{
	// Parse command line arguments
	if(argc < 6)
    {
		usage();
		exit(0);
    }
	char* imageFile = argv[1];
	char* labelFile = argv[2];
	char* paramFile = argv[3];
	char* outputFile = argv[4];
	int numRepetitions = atoi(argv[5]);

        // Initialize timers, start the runtime timer
	hipEvent_t startTime1, startTime2, stopTime1, stopTime2;
	hipEventCreate(&startTime1);
	hipEventCreate(&startTime2);
	hipEventCreate(&stopTime1);
	hipEventCreate(&stopTime2);
	float elapsedTime1, elapsedTime2;
	hipEventRecord(startTime1, 0);


        // Load image, send to GPU
	image<unsigned char>* input = loadPGM(imageFile);
	const int HEIGHT = input->height();
	const int WIDTH = input->width();
	const int SIZE = HEIGHT*WIDTH*sizeof(char);

	unsigned char* intensity = new unsigned char[numRepetitions*HEIGHT*WIDTH];
	for(int i=0; i<numRepetitions; i++)
		memcpy(&intensity[i*HEIGHT*WIDTH], input->data, SIZE);

	unsigned char* intensityDev = NULL;
	hipMalloc((void**)&intensityDev, numRepetitions*SIZE);
	hipMemcpyAsync(intensityDev, intensity, numRepetitions*SIZE, hipMemcpyHostToDevice);


        // Load connected component labels, send to GPU
	input = loadPGM(labelFile);

	unsigned char* labels = new unsigned char[numRepetitions*HEIGHT*WIDTH];
	for(int i=0; i<numRepetitions; i++)
		memcpy(&labels[i*HEIGHT*WIDTH], input->data, SIZE);

	unsigned char* labelsDev = NULL;
	hipMalloc((void **)&labelsDev, numRepetitions*SIZE);
	hipMemcpyAsync(labelsDev, labels, numRepetitions*SIZE, hipMemcpyHostToDevice);


	// Load parameters, send to GPU
	ifstream paramStream;
	paramStream.open(paramFile);

	if(paramStream.is_open() != true)
	{
		cerr << "Could not open '" << paramFile << "'." << endl;
		exit(1);
	}

	int targetLabels[1024];
	int lowerIntensityBounds[1024];
	int upperIntensityBounds[1024];

	int numLabels = 0;
	while(paramStream.eof() == false)
	{
		char line[16];
		paramStream.getline(line, 16);
		
		if(paramStream.eof() == true)
			break;

		if(numLabels % 3 == 0)
			targetLabels[numLabels/3] = strtol(line, NULL, 10);
		else if(numLabels % 3 == 1)
			lowerIntensityBounds[numLabels/3] = strtol(line, NULL, 10);
		else
			upperIntensityBounds[numLabels/3] = strtol(line, NULL, 10);
		
		numLabels++;
	}
	
	if(numLabels % 3 == 0)
		numLabels /= 3;
	else
	{
		cerr << "Number of lines in " << paramFile << " is not divisible by 3. Try '" << argv[0]
			<< " --help' for additional information." << endl;
		exit(1);
	}
	paramStream.close();

	int* targetLabelsDev = NULL;
        hipMalloc((void**)&targetLabelsDev, numLabels*sizeof(int));
        hipMemcpyAsync(targetLabelsDev, targetLabels, numLabels*sizeof(int), hipMemcpyHostToDevice);

        int* lowerIntensityBoundsDev = NULL;
        hipMalloc((void**)&lowerIntensityBoundsDev, numLabels*sizeof(int));
        hipMemcpyAsync(lowerIntensityBoundsDev, lowerIntensityBounds, numLabels*sizeof(int), hipMemcpyHostToDevice);

        int* upperIntensityBoundsDev = NULL;
        hipMalloc((void**)&upperIntensityBoundsDev, numLabels*sizeof(int));
        hipMemcpyAsync(upperIntensityBoundsDev, upperIntensityBounds, numLabels*sizeof(int), hipMemcpyHostToDevice);


        // Allocate arrays for speed and phi in GPU memory
	signed char* speedDev = NULL;
	signed char* phiDev = NULL;
	hipMalloc((void**)&speedDev, numRepetitions*numLabels*SIZE);
	hipMalloc((void**)&phiDev, numRepetitions*numLabels*SIZE);

	hipDeviceSynchronize();


	// Start the segmentation timer
	hipEventRecord(startTime2, 0);
	

	// Launch kernel to begin image segmenation
	for(int i=0; i<numRepetitions; i++)
	{
		evolveContour<<<1, numLabels>>>(intensityDev, labelsDev, speedDev, phiDev, HEIGHT, WIDTH, targetLabelsDev, i,
						numLabels, lowerIntensityBoundsDev, upperIntensityBoundsDev);
	}
	hipDeviceSynchronize();


	// Stop the segmentation timer
	hipEventRecord(stopTime2, 0);


	// Retrieve results from the GPU
	signed char* phi = new signed char[numRepetitions*numLabels*HEIGHT*WIDTH];
	hipMemcpy(phi, phiDev, numRepetitions*numLabels*SIZE, hipMemcpyDeviceToHost);


	// Stop the runtime timer
	hipEventRecord(stopTime1, 0);


	// Caio: Output: DEV
	FILE *fout;
	fout = fopen(outputFile, "wb");
	if (!fout)
	{
		printf("Could not open output file. %s\n", outputFile);
		exit(0);
	}
	fwrite(phi, numRepetitions*numLabels*SIZE, 1, fout);
	fclose(fout);
	printf("GOLD written to file.\n");


        // Stop runtime timer and print times
	hipEventElapsedTime(&elapsedTime1, startTime1, stopTime1);
	hipEventElapsedTime(&elapsedTime2, startTime2, stopTime2);
	cout << "Computation time: " << setprecision(6) << elapsedTime2 << " ms"<< endl;
	cout << "Total time: " << setprecision(6) << elapsedTime1 << " ms"<< endl;
	

	// Free resources and end the program
	hipEventDestroy(startTime1);
	hipEventDestroy(stopTime1);
	hipEventDestroy(startTime2);
	hipEventDestroy(stopTime2);

	hipFree(intensityDev);
	hipFree(labelsDev);
	hipFree(speedDev);
	hipFree(phiDev);
	hipFree(targetLabelsDev);
	hipFree(lowerIntensityBoundsDev);
	hipFree(upperIntensityBoundsDev);

        return 0;
}


image<unsigned char>* loadPGM(const char* name)
{
	char buf[BUF_SIZE];

	// Read header
	ifstream file(name, ios::in | ios::binary);
	readPNM(file, buf);
	if(strncmp(buf, "P5", 2))
	{
		cerr << "Unable to open '" << name << "'." << endl;
		throw errorPNM();
	}

	readPNM(file, buf);
	int width = atoi(buf);
	readPNM(file, buf);
	int height = atoi(buf);

	readPNM(file, buf);
	if(atoi(buf) > UCHAR_MAX)
	{
		cerr << "Unable to open '" << name << "'." << endl;
		throw errorPNM();
	}

	// Read data
	image<unsigned char>* im = new image<unsigned char>(width, height);
	file.read((char*)imPtr(im, 0, 0), width*height*sizeof(unsigned char));

	return im;
}


void readPNM(ifstream &file, char* buf)
{
	char doc[BUF_SIZE];
	char c;

	file >> c;
	while (c == '#')
	{
		file.getline(doc, BUF_SIZE);
		file >> c;
	}
	file.putback(c);

	file.width(BUF_SIZE);
	file >> buf;
	file.ignore();
}


void savePPM(image<Color>* im, const char* name)
{
	int width = im->width();
	int height = im->height();
	ofstream file(name, ios::out | ios::binary);

	file << "P6\n" << width << " " << height << "\n" << UCHAR_MAX << "\n";
	file.write((char*)imPtr(im, 0, 0), width*height*sizeof(Color));
}


Color randomColor()
{
	Color c;
	c.r = (unsigned char) rand();
	c.g = (unsigned char) rand();
	c.b = (unsigned char) rand();

	return c;
}


__global__ void evolveContour(unsigned char* intensity, unsigned char* labels, signed char* speed, signed char* phi, int HEIGHT, int WIDTH, int* targetLabels, int kernelID, int numLabels, int* lowerIntensityBounds, int* upperIntensityBounds)
{
        int tid = threadIdx.x;

        intensity = &intensity[kernelID*HEIGHT*WIDTH];
        labels = &labels[kernelID*HEIGHT*WIDTH];
        speed = &speed[(kernelID*numLabels+tid)*HEIGHT*WIDTH];
        phi = &phi[(kernelID*numLabels+tid)*HEIGHT*WIDTH];

        dim3 dimGrid(WIDTH/30+1, HEIGHT/30+1);
        dim3 dimBlock(32, 32);
        initSpeedPhi<<<dimGrid, dimBlock>>>(intensity, labels, speed, phi, HEIGHT, WIDTH, targetLabels[tid], lowerIntensityBounds[tid], upperIntensityBounds[tid]);

        int numIterations = 0;
        stopCondition[tid] = 1;
        while(stopCondition[tid])
        {
                stopCondition[tid] = 0;
                numIterations++;
                dimGrid.x = WIDTH/30+1;
                dimGrid.y = HEIGHT/30+1;
 
		// Outward evolution
                switchIn<<<dimGrid, dimBlock>>>(speed, phi, HEIGHT, WIDTH);

                // Inward evolution
                switchOut<<<dimGrid, dimBlock>>>(speed, phi, HEIGHT, WIDTH);

                // Check stopping condition on every third iteration
                if(numIterations % 3 == 0)
                {
                        dimGrid.x = WIDTH/32+1;
                        dimGrid.y = HEIGHT/32+1;
                        checkStopCondition<<<dimGrid, dimBlock>>>(speed, phi, tid, HEIGHT, WIDTH);
                        hipDeviceSynchronize();
                }
		else
			stopCondition[tid] = 1;

                if(stopCondition[tid] == 0)
                	printf("Target label %d (intensities: %d-%d) converged in %d iterations.\n", targetLabels[tid], lowerIntensityBounds[tid], upperIntensityBounds[tid], numIterations);
	}
}


__global__ void initSpeedPhi(unsigned char* intensity, unsigned char* labels, signed char* speed, signed char* phi, int HEIGHT, int WIDTH, int targetLabel, int lowerIntensityBound, int upperIntensityBound)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int xPos = 30*bx + tx;
	int yPos = 30*by + ty;

	int intensityReg;
	int speedReg;
	int phiReg;
	__shared__ int labelsTile[32][32];

	// Load data into shared memory and registers
	if(xPos < WIDTH && yPos < HEIGHT)
	{
		labelsTile[ty][tx] = labels[yPos*WIDTH+xPos];
		intensityReg = intensity[yPos*WIDTH+xPos];
	}

	// Initialization
	if(tx > 0 && tx < 31 && ty > 0 && ty < 31 && xPos < WIDTH-1 && yPos < HEIGHT-1)
	{
		// Phi
		if(labelsTile[ty][tx] != targetLabel)
		{
			if(labelsTile[ty][tx-1] != targetLabel && labelsTile[ty][tx+1] != targetLabel && labelsTile[ty-1][tx] != targetLabel && labelsTile[ty+1][tx] != targetLabel)
				phiReg = 3;
			else
				phiReg = 1;
		}
		else
		{
			if(labelsTile[ty][tx-1] != targetLabel || labelsTile[ty][tx+1] != targetLabel || labelsTile[ty-1][tx] != targetLabel || labelsTile[ty+1][tx] != targetLabel)
				phiReg = -1;
			else
				phiReg = -3;
		}

		// Speed
		if(intensityReg >= lowerIntensityBound && intensityReg <= upperIntensityBound)
			speedReg = 1;
		else
			speedReg = -1;

		// Load data back into global memory
		speed[yPos*WIDTH+xPos] = speedReg;
		phi[yPos*WIDTH+xPos] = phiReg;
	}
}


__global__ void switchIn(signed char* speed, signed char* phi, int HEIGHT, int WIDTH)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int xPos = 30*bx + tx;
	int yPos = 30*by + ty;

	int speedReg;
	__shared__ int phiTile[32][32];

	// Load data into shared memory and registers
	if(xPos < WIDTH && yPos < HEIGHT)
	{
		speedReg = speed[yPos*WIDTH+xPos];
		phiTile[ty][tx] = phi[yPos*WIDTH+xPos];
	}

	if(xPos > 0 && xPos < WIDTH-1 && yPos > 0 && yPos < HEIGHT-1)
	{
		// Delete points from Lout and add them to Lin
		if(phiTile[ty][tx] == 1 && speedReg > 0)
			phiTile[ty][tx] = -1;

		if(tx > 0 && tx < 31 && ty > 0 && ty < 31)
		{
			// Update neighborhood
			if(phiTile[ty][tx] == 3)
			{
				if(phiTile[ty][tx-1] == -1 || phiTile[ty][tx+1] == -1 || phiTile[ty-1][tx] == -1 || phiTile[ty+1][tx] == -1)
					phiTile[ty][tx] = 1;
			}

			// Eliminate redundant points in Lin
			if(phiTile[ty][tx] == -1)
			{
				if(phiTile[ty][tx-1] < 0 && phiTile[ty][tx+1] < 0 && phiTile[ty-1][tx] < 0 && phiTile[ty+1][tx] < 0)
					phiTile[ty][tx] = -3;
			}

			// Load data back into global memory
			phi[yPos*WIDTH+xPos] = phiTile[ty][tx];
		}
	}
}


__global__ void switchOut(signed char* speed, signed char* phi, int HEIGHT, int WIDTH)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int xPos = 30*bx + tx;
	int yPos = 30*by + ty;

	int speedReg;
	__shared__ int phiTile[32][32];

	// Load data into shared memory and registers
	if(xPos < WIDTH && yPos < HEIGHT)
	{
		speedReg = speed[yPos*WIDTH+xPos];
		phiTile[ty][tx] = phi[yPos*WIDTH+xPos];
	}

	if(xPos > 0 && xPos < WIDTH-1 && yPos > 0 && yPos < HEIGHT-1)
	{
		// Delete points from Lin and add them to Lout
		if(phiTile[ty][tx] == -1 && speedReg < 0)
			phiTile[ty][tx] = 1;

		if(tx > 0 && tx < 31 && ty > 0 && ty < 31)
		{
			// Update neighborhood
			if(phiTile[ty][tx] == -3)
			{
				if(phiTile[ty][tx-1] == 1 || phiTile[ty][tx+1] == 1 || phiTile[ty-1][tx] == 1 || phiTile[ty+1][tx] == 1)
					phiTile[ty][tx] = -1;
			}

			// Eliminate redundant points
			if(phiTile[ty][tx] == 1)
			{
				if(phiTile[ty][tx-1] > 0 && phiTile[ty][tx+1] > 0 && phiTile[ty-1][tx] > 0 && phiTile[ty+1][tx] > 0)
					phiTile[ty][tx] = 3;
			}

			// Load data back into global memory
			phi[yPos*WIDTH+xPos] = phiTile[ty][tx];
		}
	}

}


__global__ void checkStopCondition(signed char* speed, signed char* phi, int parentThreadID, int HEIGHT, int WIDTH)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int xPos = 32*bx + tx;
	int yPos = 32*by + ty;

	int speedReg;
	int phiReg;

	// Load data into shared memory and registers
	if(xPos < WIDTH && yPos < HEIGHT)
	{
		speedReg = speed[yPos*WIDTH+xPos];
		phiReg = phi[yPos*WIDTH+xPos];
	}

	// Falsify stop condition if criteria are not met
	if(phiReg == 1 && speedReg > 0)
		stopCondition[parentThreadID]=0;
	else if(phiReg == -1 && speedReg < 0)
		stopCondition[parentThreadID]=1;
}
